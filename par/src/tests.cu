#include "hip/hip_runtime.h"
#include "tests.h"

#include <stdio.h>
#include <stdlib.h>

#include "mnist.h"
#include "matrix.h"
#include "ann.h"
#include <math.h>
#include <string.h>
#include <time.h>

void matrix_dot_ref(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert((m1->columns == m2->rows) &&
           (m1->rows == res->rows) &&
           (m2->columns == res->columns));

    for (int row = 0; row < m1->rows; row++)
    {
        for (int col = 0; col < m2->columns; col++)
        {
            int idx = col + row * m2->columns;
            __half var = 0.0;

            for (int ii = 0; ii < m1->columns; ii++)
            {
                var += m1->m[ii + row * m1->columns] * m2->m[col + ii * m2->columns];
            }

            res->m[idx] = var;
        }
    }
}

void test_matrix_gemm()
{
    printf("----------------\n");
    printf("Dot product test\n");
    printf("----------------\n");
    unsigned n = 784, m = 30, p = 10;
    matrix_t *h_m1 = alloc_matrix(n, m);
    matrix_t *d_m1 = cuda_alloc_matrix(n, m);
    for (int i = 0; i < n * m; i++)
    {
        h_m1->m[i] = i + 1;
    }
    matrix_cudaMemcpy(d_m1, h_m1, hipMemcpyHostToDevice);
    cuda_print_matrix(d_m1, false);
    printf("\n");

    matrix_t *h_m2 = alloc_matrix(m, p);
    matrix_t *d_m2 = cuda_alloc_matrix(m, p);
    for (int i = 0; i < m * p; i++)
    {
        h_m2->m[i] = i + 1;
    }
    matrix_cudaMemcpy(d_m2, h_m2, hipMemcpyHostToDevice);
    cuda_print_matrix(d_m2, false);
    printf("\n");

    matrix_t *h_m3 = alloc_matrix(n, p);
    matrix_t *h_m3ref = alloc_matrix(n, p);
    matrix_t *d_m3 = cuda_alloc_matrix(n, p);
    for (int i = 0; i < n * p; i++)
    {
        h_m3->m[i] = 0.0;
    }
    matrix_cudaMemcpy(d_m3, h_m3, hipMemcpyHostToDevice);
    matrix_gemm(d_m1, d_m2, d_m3);
    matrix_dot_ref(h_m1, h_m2, h_m3ref);
    print_matrix(h_m3ref, false);
    printf("\n");
    cuda_print_matrix(d_m3, false);
    printf("\n");
    matrix_cudaMemcpy(h_m3, d_m3, hipMemcpyDeviceToHost);

    for (int i = 0; i < n * p; i++)
    {
        if (h_m3->m[i] != h_m3ref->m[i])
        {
            int row = (int)i / p;
            int col = i % p;
            printf("(%d  %d) got: %f, expected: %f\n", row, col, h_m3->m[i], h_m3ref->m[i]);
        }
    }

    free_matrix(h_m1);
    free_matrix(h_m2);
    free_matrix(h_m3);
    cuda_free_matrix(d_m1);
    cuda_free_matrix(d_m2);
    cuda_free_matrix(d_m3);
    printf("---\n");
    printf("OK\n");
    printf("---\n");
}

void test_hadamard_product()
{
    printf("---------------------\n");
    printf("Hadamard product test\n");
    printf("---------------------\n");
    unsigned n = 50, m = 30;
    matrix_t *h_m1 = alloc_matrix(n, m);
    matrix_t *d_m1 = cuda_alloc_matrix(n, m);
    for (int i = 0; i < n * m; i++)
    {
        h_m1->m[i] = i + 1;
    }
    matrix_cudaMemcpy(d_m1, h_m1, hipMemcpyHostToDevice);
    cuda_print_matrix(d_m1, false);

    matrix_t *h_m2 = alloc_matrix(n, m);
    matrix_t *d_m2 = cuda_alloc_matrix(n, m);
    for (int i = 0; i < n * m; i++)
    {
        h_m2->m[i] = i + 1;
    }
    matrix_cudaMemcpy(d_m2, h_m2, hipMemcpyHostToDevice);
    cuda_print_matrix(d_m2, false);

    matrix_t *h_m3 = alloc_matrix(n, m);
    matrix_t *d_m3 = cuda_alloc_matrix(n, m);
    hadamard_product(d_m1, d_m2, d_m3);
    cuda_print_matrix(d_m3, false);
    matrix_cudaMemcpy(h_m3, d_m3, hipMemcpyDeviceToHost);

    for (int i = 0; i < n * m; i++)
    {
        assert(h_m3->m[i] == (__half)((i + 1) * (i + 1)));
    }

    free_matrix(h_m1);
    free_matrix(h_m2);
    free_matrix(h_m3);
    cuda_free_matrix(d_m1);
    cuda_free_matrix(d_m2);
    cuda_free_matrix(d_m3);
    printf("---\n");
    printf("OK\n");
    printf("---\n");
}

void test_matrix_transpose()
{
    printf("--------------\n");
    printf("Transpose test\n");
    printf("--------------\n");
    unsigned n = 2;
    matrix_t *h_m1 = alloc_matrix(n, n);
    matrix_t *d_m1 = cuda_alloc_matrix(n, n);
    for (int i = 0; i < n * n; i++)
    {
        h_m1->m[i] = i + 1;
    }
    matrix_cudaMemcpy(d_m1, h_m1, hipMemcpyHostToDevice);
    cuda_print_matrix(d_m1, false);

    matrix_t *h_m2 = alloc_matrix(n, n);
    matrix_t *d_m2 = cuda_alloc_matrix(n, n);

    matrix_transpose(d_m1, d_m2);
    cuda_print_matrix(d_m2, false);
    matrix_cudaMemcpy(h_m2, d_m2, hipMemcpyDeviceToHost);

    assert(h_m2->m[0] == (__half)1.0);
    assert(h_m2->m[1] == (__half)3.0);
    assert(h_m2->m[2] == (__half)2.0);
    assert(h_m2->m[3] == (__half)4.0);

    free_matrix(h_m1);
    free_matrix(h_m2);
    cuda_free_matrix(d_m1);
    cuda_free_matrix(d_m2);
    printf("---\n");
    printf("OK\n");
    printf("---\n");
}

void test_matrix_sum()
{
    printf("--------\n");
    printf("Sum test\n");
    printf("--------\n");
    unsigned n = 50, m = 30;
    matrix_t *h_m1 = alloc_matrix(n, m);
    matrix_t *d_m1 = cuda_alloc_matrix(n, m);
    for (int i = 0; i < n * m; i++)
    {
        h_m1->m[i] = i + 1;
    }
    matrix_cudaMemcpy(d_m1, h_m1, hipMemcpyHostToDevice);
    cuda_print_matrix(d_m1, false);

    matrix_t *h_m2 = alloc_matrix(n, m);
    matrix_t *d_m2 = cuda_alloc_matrix(n, m);
    matrix_sum(d_m1, d_m1, d_m2);
    cuda_print_matrix(d_m2, false);
    matrix_cudaMemcpy(h_m2, d_m2, hipMemcpyDeviceToHost);

    for (int i = 0; i < n * m; i++)
    {
        assert(h_m2->m[i] == (__half)(2.0 * (i + 1)));
    }

    free_matrix(h_m1);
    free_matrix(h_m2);
    cuda_free_matrix(d_m1);
    cuda_free_matrix(d_m2);
    printf("---\n");
    printf("OK\n");
    printf("---\n");
}

void test_matrix_minus()
{
    printf("----------\n");
    printf("Minus test\n");
    printf("----------\n");
    unsigned n = 50, m = 30;
    matrix_t *h_m1 = alloc_matrix(n, m);
    matrix_t *d_m1 = cuda_alloc_matrix(n, m);
    for (int i = 0; i < n * m; i++)
    {
        h_m1->m[i] = i + 1;
    }
    matrix_cudaMemcpy(d_m1, h_m1, hipMemcpyHostToDevice);
    cuda_print_matrix(d_m1, false);

    matrix_t *h_m2 = alloc_matrix(n, m);
    matrix_t *d_m2 = cuda_alloc_matrix(n, m);
    matrix_minus(d_m1, d_m1, d_m2);
    cuda_print_matrix(d_m2, false);
    matrix_cudaMemcpy(h_m2, d_m2, hipMemcpyDeviceToHost);

    for (int i = 0; i < n * m; i++)
    {
        assert(h_m2->m[i] == (__half)0.0);
    }

    free_matrix(h_m1);
    free_matrix(h_m2);
    cuda_free_matrix(d_m1);
    cuda_free_matrix(d_m2);
    printf("---\n");
    printf("OK\n");
    printf("---\n");
}

int run_tests()
{
    test_matrix_gemm();
    test_hadamard_product();
    test_matrix_transpose();
    test_matrix_sum();
    test_matrix_minus();
    return 0;
}