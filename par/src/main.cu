#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "mnist.h"
#include "matrix.h"
#include "ann.h"
#include "log.h"
#include "tests.h"
#include "hip/hip_runtime_api.h"
#include "model_utils.h"

int main(int argc, char *argv[])
{
#ifdef DEBUG
    setvbuf(stdout, NULL, _IOLBF, 0);
    run_tests();
#endif
    srand(0);
    log_debug("Starting program");
    unsigned datasize, ntest;
    log_debug("Reading files");
    image *train_img = read_images("/home/charles/Developer/TP-GPGPU/train-images-idx3-ubyte", &datasize);
    byte *train_label = read_labels("/home/charles/Developer/TP-GPGPU/train-labels-idx1-ubyte", &datasize);
    image *test_img = read_images("/home/charles/Developer/TP-GPGPU/t10k-images-idx3-ubyte", &ntest);
    byte *test_label = read_labels("/home/charles/Developer/TP-GPGPU/t10k-labels-idx1-ubyte", &ntest);
    log_debug("Done reading files");

    image *d_train_img;
    byte *d_train_label;
    image *d_test_img;
    byte *d_test_label;
    hipMalloc((void **)&d_train_img, sizeof(image) * datasize);
    hipMalloc((void **)&d_train_label, sizeof(byte) * datasize);
    hipMemcpy(d_train_img, train_img, sizeof(image) * datasize, hipMemcpyHostToDevice);
    hipMemcpy(d_train_label, train_label, sizeof(byte) * datasize, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_test_img, sizeof(image) * ntest);
    hipMalloc((void **)&d_test_label, sizeof(byte) * ntest);
    hipMemcpy(d_test_img, test_img, sizeof(image) * ntest, hipMemcpyHostToDevice);
    hipMemcpy(d_test_label, test_label, sizeof(byte) * ntest, hipMemcpyHostToDevice);

    log_debug("Creating neural network");
    ann_t *nn;
    double alpha = 0.05;
    unsigned minibatch_size = 16;
    unsigned number_of_layers = 3;
    unsigned nneurons_per_layer[3] = {28 * 28, 30, 10};
    nn = create_ann(alpha, minibatch_size, number_of_layers, nneurons_per_layer);
#ifdef DEBUG
    print_nn(nn);
#endif

    log_info("starting accuracy %lf", accuracy(d_test_img, d_test_label, test_label, ntest, minibatch_size, nn));

    unsigned *shuffled_idx = (unsigned *)malloc(datasize * sizeof(unsigned));
    double *x = (double *)malloc(28 * 28 * minibatch_size * sizeof(double));
    double *y = (double *)malloc(10 * minibatch_size * sizeof(double));
    matrix_t *out = cuda_alloc_matrix(10, minibatch_size);

    for (int epoch = 0; epoch < 40; epoch++)
    {
        log_info("start learning epoch %d", epoch);

        shuffle(shuffled_idx, datasize, datasize);

        for (int i = 0; i < datasize - minibatch_size; i += minibatch_size)
        {
            gpuPopulateMinibatch(nn->layers[0]->d_activations->m, out->m, shuffled_idx + i, minibatch_size, d_train_img, 28 * 28, d_train_label, datasize);
            forward(nn);
            backward(nn, out);
        }
        log_info("epoch %d accuracy %lf", epoch, accuracy(d_test_img, d_test_label, test_label, ntest, minibatch_size, nn));
    }
    log_info("ending accuracy %lf", accuracy(d_test_img, d_test_label, test_label, ntest, minibatch_size, nn));

    free(x);
    free(y);
    free(shuffled_idx);
    cuda_free_matrix(out);

    return 0;
}
