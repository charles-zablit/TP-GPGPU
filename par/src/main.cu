#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "mnist.h"
#include "matrix.h"
#include "ann.h"
#include "log.h"
#include "tests.h"
#include "hip/hip_runtime_api.h"
#include "model_utils.h"

int main(int argc, char *argv[])
{
#ifdef DEBUG
    setvbuf(stdout, NULL, _IOLBF, 0);
    run_tests();
#endif
    srand(0);
    log_debug("Starting program");
    unsigned datasize, ntest;
    log_debug("Reading files");
    image *train_img = read_images("/home/charles/Developer/TP-GPGPU/train-images-idx3-ubyte", &datasize);
    byte *train_label = read_labels("/home/charles/Developer/TP-GPGPU/train-labels-idx1-ubyte", &datasize);
    image *test_img = read_images("/home/charles/Developer/TP-GPGPU/t10k-images-idx3-ubyte", &ntest);
    byte *test_label = read_labels("/home/charles/Developer/TP-GPGPU/t10k-labels-idx1-ubyte", &ntest);
    log_debug("Done reading files");

    log_debug("Creating neural network");
    ann_t *nn;
    double alpha = 0.05;
    unsigned minibatch_size = 16;
    unsigned number_of_layers = 3;
    unsigned nneurons_per_layer[3] = {28 * 28, 30, 10};
    nn = create_ann(alpha, minibatch_size, number_of_layers, nneurons_per_layer);
#ifdef DEBUG
    print_nn(nn);
#endif

    log_info("starting accuracy %lf", accuracy(test_img, test_label, ntest, minibatch_size, nn));

    unsigned *shuffled_idx = (unsigned *)malloc(datasize * sizeof(unsigned));
    double *x = (double *)malloc(28 * 28 * minibatch_size * sizeof(double));
    double *y = (double *)malloc(10 * minibatch_size * sizeof(double));
    matrix_t *out = cuda_alloc_matrix(10, minibatch_size);

    for (int epoch = 0; epoch < 40; epoch++)
    {
        log_info("start learning epoch %d", epoch);

        shuffle(shuffled_idx, datasize, datasize);

        for (int i = 0; i < datasize - minibatch_size; i += minibatch_size)
        {
            populate_minibatch(x, y, shuffled_idx + i, minibatch_size, train_img, 28 * 28, train_label, datasize);
            hipMemcpy(nn->layers[0]->d_activations->m, x, 28 * 28 * minibatch_size * sizeof(double), hipMemcpyHostToDevice);
            kernelRetchk;
            forward(nn);
            hipMemcpy(out->m, y, 10 * minibatch_size * sizeof(double), hipMemcpyHostToDevice);
            kernelRetchk;
            backward(nn, out);
        }
        log_info("epoch %d accuracy %lf", epoch, accuracy(test_img, test_label, ntest, minibatch_size, nn));
    }
    log_info("ending accuracy %lf", accuracy(test_img, test_label, ntest, minibatch_size, nn));

    free(x);
    free(y);
    free(shuffled_idx);
    cuda_free_matrix(out);

    return 0;
}
