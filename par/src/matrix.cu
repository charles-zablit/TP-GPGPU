#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "hip/hip_runtime_api.h"
#include "matrix.h"

matrix_t *cuda_alloc_matrix(uint16_t rows, uint16_t columns)
{
    matrix_t *g_res = (matrix_t *)malloc(sizeof(matrix_t));
    __half *m;
    hipMalloc((__half **)&m, columns * rows * sizeof(__half));
    kernelRetchk;
    g_res->m = m;
    g_res->columns = columns;
    g_res->rows = rows;
    return g_res;
}

matrix_t *alloc_matrix(uint16_t rows, uint16_t columns)
{
    matrix_t *res = (matrix_t *)malloc(sizeof(matrix_t));
    res->m = (__half *)calloc(columns * rows, sizeof(__half));
    res->columns = columns;
    res->rows = rows;
    return res;
}

void free_matrix(matrix_t *m)
{
    free(m->m);
    free(m);
}

void cuda_free_matrix(matrix_t *m)
{
    hipFree(m->m);
    kernelRetchk;
    free(m);
}

void print_matrix(matrix_t *m, bool is_short)
{
    uint16_t lim_rows = 0;
    uint16_t lim_col = 0;

    if (is_short)
    {
        lim_rows = MIN(m->rows, 4);
        lim_col = MIN(m->columns, 10);
    }
    else
    {
        lim_rows = m->rows;
        lim_col = m->columns;
    }

    for (int row = 0; row < lim_rows; row++)
    {
        printf("|");
        for (int col = 0; col < lim_col; col++)
        {
            printf("%.2lf ", __half2float(m->m[col + row * m->columns]));
        }
        if (is_short && lim_col != m->columns)
            printf("...");
        printf("|\n");
    }
    if (is_short && lim_rows != m->rows)
    {
        printf("...\n");
    }
}

void cuda_print_matrix(matrix_t *d_m, bool is_short)
{
    matrix_t *m = alloc_matrix(d_m->rows, d_m->columns);
    matrix_cudaMemcpy(m, d_m, hipMemcpyDeviceToHost);
    print_matrix(m, is_short);
    free_matrix(m);
}

__global__ void hadamard_product_kernel(__half *A, __half *B, __half *C, uint16_t numRows, uint16_t numColumns)
{
    uint16_t row = blockIdx.y * blockDim.y + threadIdx.y;
    uint16_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numRows && col < numColumns)
    {
        C[row * numColumns + col] = A[row * numColumns + col] * B[row * numColumns + col];
    }
}

void hadamard_product(matrix_t *d_m1, matrix_t *d_m2, matrix_t *d_res)
{
    assert((d_m1->columns == d_m2->columns) &&
           (d_m1->columns == d_res->columns) &&
           (d_m1->rows == d_m2->rows) &&
           (d_m1->rows == d_res->rows));

    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 blocksPerGrid((d_m1->columns + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (d_m1->rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hadamard_product_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_m1->m, d_m2->m, d_res->m, d_res->rows, d_res->columns);
    kernelRetchk;
}

__global__ void matrix_minus_kernel(__half *A, __half *B, __half *C, uint16_t numRows, uint16_t numColumns)
{
    uint16_t row = blockIdx.y * blockDim.y + threadIdx.y;
    uint16_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numRows && col < numColumns)
    {
        C[row * numColumns + col] = A[row * numColumns + col] - B[row * numColumns + col];
    }
}

void matrix_minus(matrix_t *d_m1, matrix_t *d_m2, matrix_t *d_res)
{
    assert((d_m1->columns == d_m2->columns) &&
           (d_m1->columns == d_res->columns) &&
           (d_m1->rows == d_m2->rows) &&
           (d_m1->rows == d_res->rows));

    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 blocksPerGrid((d_m1->columns + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (d_m1->rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_minus_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_m1->m, d_m2->m, d_res->m, d_res->rows, d_res->columns);
    kernelRetchk;
}

void matrix_gemm(hipblasHandle_t *handle, matrix_t *d_m1, matrix_t *d_m2, matrix_t *d_res, hipblasOperation_t t_m1, hipblasOperation_t t_m2, __half alpha, __half beta)
{
    const int m = t_m1 == HIPBLAS_OP_T ? d_m1->columns : d_m1->rows;
    const int n = t_m2 == HIPBLAS_OP_T ? d_m2->rows : d_m2->columns;
    const int k = t_m1 == HIPBLAS_OP_T ? d_m1->rows : d_m1->columns;

    assert((m == d_res->rows) &&
           (n == d_res->columns));

    CUBLAS_CHECK(hipblasHgemm(*handle,
                             t_m2,
                             t_m1,
                             n, m, k,
                             &alpha,
                             d_m2->m, d_res->columns,
                             d_m1->m, d_m1->columns,
                             &beta,
                             d_res->m, d_res->columns));
}

__global__ void matrix_function_kernel(__half *A, __half *B, bool prime, uint16_t numRows, uint16_t numColumns)
{
    const uint16_t row = blockIdx.y * blockDim.y + threadIdx.y;
    const uint16_t col = blockIdx.x * blockDim.x + threadIdx.x;
    const __half one = __float2half(1.0f);
    if (row < numRows && col < numColumns)
    {
        __half x = A[row * numColumns + col];
        __half sig = one / (one + hexp(-x));
        if (prime)
        {
            sig = sig * (one - sig);
        }
        B[row * numColumns + col] = sig;
    }
}

void matrix_function(matrix_t *d_m, bool prime, matrix_t *d_res)
{
    assert((d_m->columns == d_res->columns) &&
           (d_m->rows == d_res->rows));

    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 blocksPerGrid((d_m->columns + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (d_m->rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_function_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_m->m, d_res->m, prime, d_res->rows, d_res->columns);
    kernelRetchk;
}

__global__ void matrix_transpose_kernel(__half *A, __half *B, uint16_t nb_rows, uint16_t nb_cols)
{
    __shared__ __half s[THREADS_PER_BLOCK][THREADS_PER_BLOCK + 1];

    uint16_t row = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
    uint16_t col = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
    if ((row < nb_rows) && (col < nb_cols))
    {
        s[threadIdx.y][threadIdx.x] = A[col * nb_rows + row];
    }

    __syncthreads();

    row = blockIdx.y * THREADS_PER_BLOCK + threadIdx.x;
    col = blockIdx.x * THREADS_PER_BLOCK + threadIdx.y;
    if ((row < nb_cols) && (col < nb_rows))
    {
        B[col * nb_cols + row] = s[threadIdx.x][threadIdx.y];
    }
}

void matrix_transpose(matrix_t *d_m, matrix_t *d_res)
{
    assert((d_m->columns == d_res->rows) &&
           (d_m->rows == d_res->columns));

    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 blocksPerGrid((d_m->columns + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (d_m->rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_m->m, d_res->m, d_res->rows, d_res->columns);
    kernelRetchk;
}

void matrix_memcpy(matrix_t *dest, const matrix_t *src)
{
    assert((dest->rows == src->rows) &&
           (dest->columns == src->columns));

    memcpy(dest->m, src->m, src->columns * src->rows * sizeof(__half));
}

void matrix_cudaMemcpy(matrix_t *dest, const matrix_t *src, hipMemcpyKind kind)
{
    assert((dest->rows == src->rows) &&
           (dest->columns == src->columns));

    hipMemcpy(dest->m, src->m, src->columns * src->rows * sizeof(__half), kind);
    kernelRetchk;
}

void init_ones(matrix_t *d_m)
{
    matrix_t *h_m = alloc_matrix(d_m->rows, d_m->columns);
    for (int idx = 0; idx < h_m->columns * h_m->rows; idx++)
    {
        h_m->m[idx] = __float2half(1.0f);
    }
    matrix_cudaMemcpy(d_m, h_m, hipMemcpyHostToDevice);
}